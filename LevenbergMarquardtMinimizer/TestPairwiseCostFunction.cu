#include "hip/hip_runtime.h"
#include "PairwiseCostFunctionAndItsGradientWithRespectToParams.h"
#include "TestPairwiseCostFunction.h"
#include <algorithm>
#include <cusp/array1d.h>
#include <cusp/array2d.h>
#include <cusp/csr_matrix.h>
#include <cusp/elementwise.h>
#include <cusp/multiply.h>
#include <cusp/transpose.h>
#include <cusp/print.h>
#include <iostream>
#include <thrust/device_ptr.h>

void testPairwiseCostFunction(size_t numPoints, size_t maxPoints/*= 65535*/)
{
  const size_t numDims = 3;
  const size_t numParams = 12;

  typedef cusp::array1d<float, cusp::host_memory> HostArray1d;
  typedef cusp::array2d<float, cusp::host_memory, cusp::row_major> HostArray2d;

  HostArray2d tildePi(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    tildePi(i, 0) = 1;
    tildePi(i, 1) = 0;
    tildePi(i, 2) = 0;
  }

  HostArray2d si(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    si(i, 0) = 0;
    si(i, 1) = 0;
    si(i, 2) = 0;
  }

  HostArray2d ti(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    ti(i, 0) = 1;
    ti(i, 1) = 1;
    ti(i, 2) = 1;
  }

  HostArray2d tildePj(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    tildePj(i, 0) = 0;
    tildePj(i, 1) = 1;
    tildePj(i, 2) = 0;
  }

  HostArray2d sj(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    sj(i, 0) = 0.5;
    sj(i, 1) = 0;
    sj(i, 2) = 0;
  }

  HostArray2d tj(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    tj(i, 0) = 1;
    tj(i, 1) = 1;
    tj(i, 2) = 1;
  }

  typedef cusp::array1d<float, cusp::device_memory> DeviceArray1d;
  typedef cusp::array2d<float, cusp::device_memory, cusp::row_major> DeviceArray2d;

  DeviceArray2d deviceTildePi(tildePi);
  DeviceArray2d deviceSi(si);
  DeviceArray2d deviceTi(ti);

  DeviceArray2d deviceTildePj(tildePj);
  DeviceArray2d deviceSj(sj);
  DeviceArray2d deviceTj(tj);

  HostArray2d jacTildePi(numPoints, numDims * numParams, 0);

  HostArray2d jacSi(numPoints, numDims * numParams, 0);

  for (size_t i = 0; i < numPoints; ++i)
  {
    cusp::array2d_view<HostArray2d::row_view, cusp::row_major> jac(numDims, numParams, numParams, jacSi.row(i));

    jac(0, 0) = 1;
    jac(1, 1) = 1;
    jac(2, 2) = 1;
  }

  HostArray2d jacTi(numPoints, numDims * numParams, 0);

  for (size_t i = 0; i < numPoints; ++i)
  {
    cusp::array2d_view<HostArray2d::row_view, cusp::row_major> jac(numDims, numParams, numParams, jacTi.row(i));

    jac(0, 3) = 1;
    jac(1, 4) = 1;
    jac(2, 5) = 1;
  }

  HostArray2d jacTildePj(numPoints, numDims * numParams, 0);

  HostArray2d jacSj(numPoints, numDims * numParams, 0);

  for (size_t i = 0; i < numPoints; ++i)
  {
    cusp::array2d_view<HostArray2d::row_view, cusp::row_major> jac(numDims, numParams, numParams, jacSj.row(i));

    jac(0, 6) = 1;
    jac(1, 7) = 1;
    jac(2, 8) = 1;
  }

  HostArray2d jacTj(numPoints, numDims * numParams, 0);

  for (size_t i = 0; i < numPoints; ++i)
  {
    cusp::array2d_view<HostArray2d::row_view, cusp::row_major> jac(numDims, numParams, numParams, jacTj.row(i));

    jac(0, 9) = 1;
    jac(1, 10) = 1;
    jac(2, 11) = 1;
  }

  DeviceArray2d deviceJacTildePi(jacTildePi);
  DeviceArray2d deviceJacSi(jacSi);
  DeviceArray2d deviceJacTi(jacTi);

  DeviceArray2d deviceJacTildePj(jacTildePj);
  DeviceArray2d deviceJacSj(jacSj);
  DeviceArray2d deviceJacTj(jacTj);

  hipEvent_t start;
  hipEventCreate(&start);

  hipEvent_t stop;
  hipEventCreate(&stop);

  DeviceArray1d ei(numPoints);
  DeviceArray1d ej(numPoints);

  const size_t numCols = numPoints * numParams;

  HostArray1d columnIndicesi(numCols);
  HostArray1d columnIndicesj(numCols);

  for (int i = 0; i < numCols; ++i)
  {
    columnIndicesi[i] = i;
    if ((i % numParams) < (numParams / 2))
    {
      columnIndicesj[i + numParams / 2] = i;
    }
    else
    {
      columnIndicesj[i - numParams / 2] = i;
    }
  }

  HostArray1d rowOffsetsi(numPoints + 1);
  HostArray1d rowOffsetsj(numPoints + 1);

  for (int i = 0; i <= numPoints; ++i)
  {
    rowOffsetsi[i] = i * numParams;
    rowOffsetsj[i] = i * numParams;
  }

  typedef cusp::csr_matrix<int, float, cusp::device_memory> DeviceCsrMatrix;

  DeviceCsrMatrix jacEi(numPoints, numCols, numPoints * numParams);
  jacEi.row_offsets = rowOffsetsi;
  jacEi.column_indices = columnIndicesi;

  DeviceCsrMatrix jacEj(numPoints, numCols, numPoints * numParams);
  jacEj.row_offsets = rowOffsetsj;
  jacEj.column_indices = columnIndicesj;

  hipEventRecord(start, 0);

  for (int i = 0; i < numPoints; i += maxPoints)
  {
    float* pTildePi = thrust::raw_pointer_cast(&deviceTildePi(i, 0));
    float* pSi = thrust::raw_pointer_cast(&deviceSi(i, 0));
    float* pTi = thrust::raw_pointer_cast(&deviceTi(i, 0));

    float* pTildePj = thrust::raw_pointer_cast(&deviceTildePj(i, 0));
    float* pSj = thrust::raw_pointer_cast(&deviceSj(i, 0));
    float* pTj = thrust::raw_pointer_cast(&deviceTj(i, 0));

    float* pJacTildePi = thrust::raw_pointer_cast(&deviceJacTildePi(i, 0));
    float* pJacSi = thrust::raw_pointer_cast(&deviceJacSi(i, 0));
    float* pJacTi = thrust::raw_pointer_cast(&deviceJacTi(i, 0));

    float* pJacTildePj = thrust::raw_pointer_cast(&deviceJacTildePj(i, 0));
    float* pJacSj = thrust::raw_pointer_cast(&deviceJacSj(i, 0));
    float* pJacTj = thrust::raw_pointer_cast(&deviceJacTj(i, 0));

    float* pEi = thrust::raw_pointer_cast(&ei[i]);
    float* pJacEi = thrust::raw_pointer_cast(&jacEi.values[i * numParams]);

    float* pEj = thrust::raw_pointer_cast(&ej[i]);
    float* pJacEj = thrust::raw_pointer_cast(&jacEj.values[i * numParams]);

    PairwiseCostFunctionAndItsGradientWithRespectToParams3x12(
      pTildePi, pSi, pTi, pJacTildePi, pJacSi, pJacTi,
      pTildePj, pSj, pTj, pJacTildePj, pJacSj, pJacTj,
      pEi, pEj, pJacEi, pJacEj, std::min(numPoints - i, maxPoints));
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float timeElapsedMs;
  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for the kernel <PairwiseCostFunctionAndItsGradientWithRespectToParams3x12> " << timeElapsedMs << " ms" << std::endl;

  DeviceCsrMatrix jacEit;

  hipEventRecord(start, 0);

  cusp::transpose(jacEi, jacEit);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for transpose(jacEi) " << timeElapsedMs << " ms" << std::endl;

  DeviceCsrMatrix jacEjt;

  hipEventRecord(start, 0);

  cusp::transpose(jacEj, jacEjt);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for transpose(jacEj) " << timeElapsedMs << " ms" << std::endl;

  DeviceArray1d jacEitTimesEi(jacEit.num_rows);

  hipEventRecord(start, 0);
  
  cusp::multiply(jacEit, ei, jacEitTimesEi);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for multiply(jacEit, ei) " << timeElapsedMs << " ms" << std::endl;

  DeviceArray1d jacEjtTimesEj(jacEjt.num_rows);

  hipEventRecord(start, 0);

  cusp::multiply(jacEjt, ej, jacEjtTimesEj);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for multiply(jacEjt, ej) " << timeElapsedMs << " ms" << std::endl;

  DeviceArray1d jacETimesE;

  hipEventRecord(start, 0);

  DeviceCsrMatrix jacEitTimesJacEi;

  hipEventRecord(start, 0);

  cusp::multiply(jacEit, jacEi, jacEitTimesJacEi);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for multiply(jacEit, jacEi) " << timeElapsedMs << " ms" << std::endl;

  DeviceCsrMatrix jacEjtTimesJacEj;

  hipEventRecord(start, 0);

  cusp::multiply(jacEjt, jacEj, jacEjtTimesJacEj);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for multiply(jacEjt, jacEj) " << timeElapsedMs << " ms" << std::endl;

  DeviceCsrMatrix jacEtTimesJacE;

  hipEventRecord(start, 0);

  cusp::add(jacEitTimesJacEi, jacEjtTimesJacEj, jacEtTimesJacE);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for add(jacEitTimesJacEi, jacEjtTimesJacEj) " << timeElapsedMs << " ms" << std::endl;
}