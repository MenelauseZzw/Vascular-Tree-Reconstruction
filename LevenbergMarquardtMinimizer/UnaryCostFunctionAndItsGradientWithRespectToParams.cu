#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ProjectionOntoLineAndItsJacobian.cuh"
#include "UnaryCostFunctionAndItsGradientWithRespectToParams.h"

__constant__ static float jacTildeP[][3] =
{
  0, 0, 0,
  0, 0, 0,
  0, 0, 0,
  0, 0, 0,
  0, 0, 0,
  0, 0, 0
};

__constant__ static float jacS[][3] =
{
  1, 0, 0,
  0, 1, 0,
  0, 0, 1,
  0, 0, 0,
  0, 0, 0,
  0, 0, 0
};

__constant__ static float jacT[][3] =
{
  0, 0, 0,
  0, 0, 0,
  0, 0, 0,
  1, 0, 0,
  0, 1, 0,
  0, 0, 1
};

template<int numDims>
__device__ void UnaryCostFunctionAndItsGradientWithRespectToParamsAt(const float* tildeP, const float* p, const float* jacTildeP, const float* jacP, float* pUnaryCostFunction, float* pUnaryCostGradient)
{
  float pMinusTildeP[numDims];
  float pMinusTildePSq = 0;

  for (int i = 0; i < numDims; ++i)
  {
    pMinusTildeP[i] = p[i] - tildeP[i];
    pMinusTildePSq += pMinusTildeP[i] * pMinusTildeP[i];
  }

  float invPMinusTildeP = rsqrtf(pMinusTildePSq);
  float nablaPMinusTildeP = 0;

  for (int i = 0; i < numDims; ++i)
  {
    nablaPMinusTildeP += pMinusTildeP[i] * (jacP[i] - jacTildeP[i]) * invPMinusTildeP;
  }

  *pUnaryCostFunction = sqrtf(pMinusTildePSq);
  *pUnaryCostGradient = nablaPMinusTildeP;
}

template<int numDims>
__global__ void UnaryCostFunctionAndItsGradientWithRespectToParams(const float* pTildeP, const float* pS, const float *pT,/* const float *pJacTildeP, const float *pJacS, const float *pJacT,*/ const float *pSigma, float* pUnaryCostFunction, float* pUnaryCostGradient)
{
  float tildeP[numDims];
  float s[numDims];
  float t[numDims];
  float p[numDims];

  const int numParams = blockDim.x;

  const int numPnt = blockIdx.x;
  const int numPar = threadIdx.x;

  const int indPnt0 = numDims * numPnt;

  for (int i = 0, indPnt = indPnt0; i < numDims; ++i, ++indPnt)
  {
    tildeP[i] = pTildeP[indPnt];
    s[i] = pS[indPnt];
    t[i] = pT[indPnt];
  }

  //float jacTildeP[numDims];
  //float jacS[numDims];
  //float jacT[numDims];
  float jacP[numDims];

  //const int indJac0 = numDims * numParams * numPnt;

  //for (int i = 0, indJac = indJac0 + numPar; i < numDims; ++i, indJac += numParams)
  //{
  //  jacTildeP[i] = pJacTildeP[indJac];
  //  jacS[i] = pJacS[indJac];
  //  jacT[i] = pJacT[indJac];
  //}

  //const int indJac0 = 0;

  //for (int i = 0, indJac = indJac0 + numPar; i < numDims; ++i, indJac += numParams)
  //{
  //  jacTildeP[i] = JacTildeP[indJac];
  //  jacS[i] = JacS[indJac];
  //  jacT[i] = JacT[indJac];
  //}

  ProjectionOntoLineAndItsJacobianAt<numDims>(tildeP, s, t, jacTildeP[numPar], jacS[numPar], jacT[numPar], p, jacP);

  const int indGrad0 = numParams * numPnt;

  float costFunction;
  float costGradient;
  UnaryCostFunctionAndItsGradientWithRespectToParamsAt<numDims>(tildeP, p, jacTildeP[numPar], jacP, &costFunction, &costGradient);

  if (!isfinite(costGradient))
  {
    const float h = 1e-6;

    float pMinusTildeP[numDims];
    float pMinusTildePSq = 0;

    if (numPar < numDims)
    {
      s[numPar % numDims] = pS[indPnt0 + numPar % numDims] + 2 * h;
    }
    else
    {
      t[numPar % numDims] = pT[indPnt0 + numPar % numDims] + 2 * h;
    }

    ProjectionOntoLineAndItsJacobianAt<numDims>(tildeP, s, t, jacTildeP[numPar], jacS[numPar], jacT[numPar], p, jacP);

    for (int i = 0; i < numDims; ++i)
    {
      pMinusTildeP[i] = p[i] - tildeP[i];
      pMinusTildePSq += pMinusTildeP[i] * pMinusTildeP[i];
    }

    float pPlusTwoHMinusTildeP = sqrtf(pMinusTildePSq);

    if (numPar < numDims)
    {
      s[numPar % numDims] = pS[indPnt0 + numPar % numDims] + h;
    }
    else
    {
      t[numPar % numDims] = pT[indPnt0 + numPar % numDims] + h;
    }

    ProjectionOntoLineAndItsJacobianAt<numDims>(tildeP, s, t, jacTildeP[numPar], jacS[numPar], jacT[numPar], p, jacP);

    for (int i = 0; i < numDims; ++i)
    {
      pMinusTildeP[i] = p[i] - tildeP[i];
      pMinusTildePSq += pMinusTildeP[i] * pMinusTildeP[i];
    }

    float pPlusHMinusTildeP = sqrtf(pMinusTildePSq);

    if (numPar < numDims)
    {
      s[numPar % numDims] = pS[indPnt0 + numPar % numDims] - h;
    }
    else
    {
      t[numPar % numDims] = pT[indPnt0 + numPar % numDims] - h;
    }

    ProjectionOntoLineAndItsJacobianAt<numDims>(tildeP, s, t, jacTildeP[numPar], jacS[numPar], jacT[numPar], p, jacP);

    for (int i = 0; i < numDims; ++i)
    {
      pMinusTildeP[i] = p[i] - tildeP[i];
      pMinusTildePSq += pMinusTildeP[i] * pMinusTildeP[i];
    }

    float pMinusHMinusTildeP = sqrtf(pMinusTildePSq);

    if (numPar < numDims)
    {
      s[numPar % numDims] = pS[indPnt0 + numPar % numDims] - 2 * h;
    }
    else
    {
      t[numPar % numDims] = pT[indPnt0 + numPar % numDims] - 2 * h;
    }

    ProjectionOntoLineAndItsJacobianAt<numDims>(tildeP, s, t, jacTildeP[numPar], jacS[numPar], jacT[numPar], p, jacP);

    for (int i = 0; i < numDims; ++i)
    {
      pMinusTildeP[i] = p[i] - tildeP[i];
      pMinusTildePSq += pMinusTildeP[i] * pMinusTildeP[i];
    }

    float pMinusTwoHMinusTildeP = sqrtf(pMinusTildePSq);

    costGradient = (-pPlusTwoHMinusTildeP + 8 * pPlusHMinusTildeP - 8 * pMinusHMinusTildeP + pMinusTwoHMinusTildeP) / (12 * h);
  }

  const float sigma = pSigma[numPnt];

  pUnaryCostFunction[numPnt] = costFunction / sigma;
  pUnaryCostGradient[indGrad0 + numPar] = costGradient / sigma;

  assert(isfinite(costFunction));
  assert(isfinite(costGradient));
}

extern "C" void UnaryCostFunctionAndItsGradientWithRespectToParams3x6(const float* pTildeP, const float* pS, const float *pT, /*const float *pJacTildeP, const float *pJacS, const float *pJacT,*/ const float *pSigma, float* pUnaryCostFunction, float* pUnaryCostGradient, int numPoints)
{
  UnaryCostFunctionAndItsGradientWithRespectToParams<3> << <numPoints, 6 >> >(pTildeP, pS, pT, /*pJacTildeP, pJacS, pJacT,*/ pSigma, pUnaryCostFunction, pUnaryCostGradient);
}