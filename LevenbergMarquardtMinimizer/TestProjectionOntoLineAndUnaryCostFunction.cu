#include "hip/hip_runtime.h"
#include "ProjectionOntoLineAndItsJacobian.h"
#include "TestProjectionOntoLineAndUnaryCostFunction.h"
#include "UnaryCostFunctionAndItsGradientWithRespectToParams.h"
#include <algorithm>
#include <cusp/array1d.h>
#include <cusp/array2d.h>
#include <cusp/print.h>
#include <iostream>
#include <thrust/device_ptr.h>

void testProjectionOntoLineAndUnaryCostFunction(size_t numPoints, size_t maxPoints/*= 65535*/)
{
  const size_t numDims = 3;
  const size_t numParams = 6;

  typedef cusp::array1d<float, cusp::host_memory> HostArray1d;
  typedef cusp::array2d<float, cusp::host_memory, cusp::row_major> HostArray2d;

  HostArray2d tildeP(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    tildeP(i, 0) = 1;
    tildeP(i, 1) = 0;
    tildeP(i, 2) = 0;
  }

  HostArray2d s(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    s(i, 0) = 0;
    s(i, 1) = 0;
    s(i, 2) = 0;
  }

  HostArray2d t(numPoints, numDims);

  for (size_t i = 0; i < numPoints; ++i)
  {
    t(i, 0) = 1;
    t(i, 1) = 1;
    t(i, 2) = 1;
  }

  typedef cusp::array1d<float, cusp::device_memory> DeviceArray1d;
  typedef cusp::array2d<float, cusp::device_memory, cusp::row_major> DeviceArray2d;

  DeviceArray2d deviceTildeP(tildeP);
  DeviceArray2d deviceS(s);
  DeviceArray2d deviceT(t);
  DeviceArray2d deviceP(numPoints, numDims);

  HostArray2d jacTildeP(numPoints, numDims * numParams, 0);

  HostArray2d jacS(numPoints, numDims * numParams, 0);

  for (size_t i = 0; i < numPoints; ++i)
  {
    cusp::array2d_view<HostArray2d::row_view, cusp::row_major> jac(numDims, numParams, numParams, jacS.row(i));

    jac(0, 0) = 1;
    jac(1, 1) = 1;
    jac(2, 2) = 1;
  }

  HostArray2d jacT(numPoints, numDims * numParams, 0);

  for (size_t i = 0; i < numPoints; ++i)
  {
    cusp::array2d_view<HostArray2d::row_view, cusp::row_major> jac(numDims, numParams, numParams, jacT.row(i));

    jac(0, 3) = 1;
    jac(1, 4) = 1;
    jac(2, 5) = 1;
  }

  DeviceArray2d deviceJacTildeP(jacTildeP);
  DeviceArray2d deviceJacS(jacS);
  DeviceArray2d deviceJacT(jacT);
  DeviceArray2d deviceJacP(numPoints, numDims * numParams);

  hipEvent_t start;
  hipEventCreate(&start);

  hipEvent_t stop;
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);

  for (int i = 0; i < numPoints; i += maxPoints)
  {
    float* pTildeP = thrust::raw_pointer_cast(&deviceTildeP(i, 0));
    float* pS = thrust::raw_pointer_cast(&deviceS(i, 0));
    float* pT = thrust::raw_pointer_cast(&deviceT(i, 0));
    float* pP = thrust::raw_pointer_cast(&deviceP(i, 0));

    float* pJacTildeP = thrust::raw_pointer_cast(&deviceJacTildeP(i, 0));
    float* pJacS = thrust::raw_pointer_cast(&deviceJacS(i, 0));
    float* pJacT = thrust::raw_pointer_cast(&deviceJacT(i, 0));
    float* pJacP = thrust::raw_pointer_cast(&deviceJacP(i, 0));

    ProjectionOntoLineAndItsJacobian3x6(pTildeP, pS, pT, pJacTildeP, pJacS, pJacT, pP, pJacP, std::min(numPoints - i, maxPoints));
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float timeElapsedMs;
  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for the kernel <ProjectionOntoLineAndItsJacobian3x6> " << timeElapsedMs << " ms" << std::endl;

  DeviceArray1d deviceUnaryCostFunction(numPoints);
  DeviceArray2d deviceUnaryCostGradient(numPoints, numParams);

  hipEventRecord(start, 0);

  for (int i = 0; i < numPoints; i += maxPoints)
  {
    float* pTildeP = thrust::raw_pointer_cast(&deviceTildeP(i, 0));
    float* pS = thrust::raw_pointer_cast(&deviceS(i, 0));
    float* pT = thrust::raw_pointer_cast(&deviceT(i, 0));
    float* pP = thrust::raw_pointer_cast(&deviceP(i, 0));

    float* pJacTildeP = thrust::raw_pointer_cast(&deviceJacTildeP(i, 0));
    float* pJacS = thrust::raw_pointer_cast(&deviceJacS(i, 0));
    float* pJacT = thrust::raw_pointer_cast(&deviceJacT(i, 0));
    float* pJacP = thrust::raw_pointer_cast(&deviceJacP(i, 0));

    float* pUnaryCostFunction = thrust::raw_pointer_cast(&deviceUnaryCostFunction[i]);
    float* pUnaryCostGradient = thrust::raw_pointer_cast(&deviceUnaryCostGradient(i, 0));

    UnaryCostFunctionAndItsGradientWithRespectToParams3x6(pTildeP, pS, pT, pJacTildeP, pJacS, pJacT, pUnaryCostFunction, pUnaryCostGradient, std::min(numPoints - i, maxPoints));
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&timeElapsedMs, start, stop);
  std::cout << "Time for the kernel <UnaryCostFunctionAndItsGradientWithRespectToParams3x6> " << timeElapsedMs << " ms" << std::endl;

  HostArray1d p(deviceP.row(numPoints - 1));
  std::cout << "p" << std::endl;
  cusp::print(p);

  HostArray2d jacP(
    cusp::array2d_view<DeviceArray2d::row_view, cusp::row_major>(numDims, numParams, numParams, deviceJacP.row(numPoints - 1))
    );

  std::cout << "jacP" << std::endl;
  cusp::print(jacP);

  float unaryCostFunction = deviceUnaryCostFunction[numPoints - 1];
  std::cout << "Unary cost function " << unaryCostFunction << std::endl;

  HostArray1d unaryCostGradient = deviceUnaryCostGradient.row(numPoints - 1);
  std::cout << "Unary cost gradient " << std::endl;
  cusp::print(unaryCostGradient);
}
