#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ProjectionOntoLineAndItsJacobian.cuh"

template<int numDims>
__global__ void ProjectionOntoLineAndItsJacobian(const float* pTildeP, const float* pS, const float *pT, const float *pJacTildeP, const float *pJacS, const float *pJacT, float *pP, float* pJacP)
{
  float tildeP[numDims];
  float s[numDims];
  float t[numDims];
  float p[numDims];

  const int numParams = blockDim.x;

  const int numPnt = blockIdx.x;
  const int numPar = threadIdx.x;

  const int indPnt0 = numDims * numPnt;

  for (int i = 0, indPnt = indPnt0; i < numDims; ++i, ++indPnt)
  {
    tildeP[i] = pTildeP[indPnt];
    s[i] = pS[indPnt];
    t[i] = pT[indPnt];
  }

  float jacTildeP[numDims];
  float jacS[numDims];
  float jacT[numDims];
  float jacP[numDims];

  const int indJac0 = numDims * numParams * numPnt;

  for (int i = 0, indJac = indJac0 + numPar; i < numDims; ++i, indJac += numParams)
  {
    jacTildeP[i] = pJacTildeP[indJac];
    jacS[i] = pJacS[indJac];
    jacT[i] = pJacT[indJac];
  }

  ProjectionOntoLineAndItsJacobianAt<numDims>(tildeP, s, t, jacTildeP, jacS, jacT, p, jacP);

  for (int i = 0, indPnt = indPnt0; i < numDims; ++i, ++indPnt)
  {
    pP[indPnt] = p[i];
  }

  for (int i = 0, indJac = indJac0 + numPar; i < numDims; ++i, indJac += numParams)
  {
    pJacP[indJac] = jacP[i];
  }
}

extern "C" void ProjectionOntoLineAndItsJacobian3x6(const float* pTildeP, const float* pS, const float *pT, const float *pJacTildeP, const float *pJacS, const float *pJacT, float *pP, float* pJacP, unsigned int numPoints)
{
  ProjectionOntoLineAndItsJacobian<3><<<numPoints, 6>>>(pTildeP, pS, pT, pJacTildeP, pJacS, pJacT, pP, pJacP);
}
